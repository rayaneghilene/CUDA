
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>



/////////// PARTIE 1 - PRISE EN MAIN DE CUDA : Multiplication de matrices ///////////




////////////////////////////////////// MATRIX INIT //////////////////////////////////////

void MatrixInit(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            M[i * p + j] = (static_cast<float>(rand()) / RAND_MAX) * 2 - 1;
}

////////////////////////////////////// MATRIX PRINT //////////////////////////////////////


void MatrixPrint(float *M, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++) {
        printf("\n");
        for (j = 0; j < p; j++)
            printf("%f\t", M[i * p + j]);
    
    }
    printf("\n");
}

////////////////////////////////////// MATRIX ADD //////////////////////////////////////


void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i, j;
    for (i = 0; i < n; i++)
        for (j = 0; j < p; j++)
            Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
}

////////////////////////////////////// MATRIX MULTIPLICATION //////////////////////////////////////


void MatrixMult(float *M1, float *M2, float *Mout, int n){
    int i, j, k;
    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++) {
            Mout[i * n + j] = 0;
            for (k = 0; k < n; k++)
                Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
        }
    
}

////////////////////////////////////// MATRIX ADD GPU //////////////////////////////////////



__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
}


////////////////////////////////////// MATRIX MULTIPLICATION GPU //////////////////////////////////////

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k;
    Mout[i * n + j] = 0;
    for (k = 0; k < n; k++)
        Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
}



int CPU_test() {
    // MatrixAdd test
    int n = 1500, p = 1500;
    float *M1, *M2, *Mout;

    M1 = (float *)malloc(n * p * sizeof(float));
    M2 = (float *)malloc(n * p * sizeof(float));

    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);

 
    Mout = (float *)malloc(n * p * sizeof(float));
    
    
    // Measure execution time for MatrixAdd
    clock_t start_time_add = clock();
    MatrixAdd(M1, M2, Mout, n, p);
    clock_t end_time_add = clock();

    double execution_time_add = ((double)end_time_add - start_time_add) / CLOCKS_PER_SEC;
    printf("Execution time for MatrixAdd: %f seconds\n", execution_time_add);


    free(M1);
    free(M2);
    free(Mout);

    // MatrixMult test
    n = 1500;
    float *M3, *M4, *Mout2;

    M3 = (float *)malloc(n * n * sizeof(float));
    M4 = (float *)malloc(n * n * sizeof(float));

    MatrixInit(M3, n, n);
    MatrixInit(M4, n, n);

    Mout2 = (float *)malloc(n * n * sizeof(float));

    // Measure execution time for MatrixMult
    clock_t start_time_mult = clock();
    MatrixMult(M3, M4, Mout2, n);
    clock_t end_time_mult = clock();

    double execution_time_mult = ((double)end_time_mult - start_time_mult) / CLOCKS_PER_SEC;
    printf("Execution time for MatrixMult: %f seconds\n", execution_time_mult);

    free(M3);
    free(M4);
    free(Mout2);

    return 0;
}



 
int GPUtest() {
    //cudaMatrixAdd test
    int n = 100, p = 100;
    float *M1, *M2, *Mout;
    float *d_M1, *d_M2, *d_Mout;

    M1 = (float *)malloc(n * p * sizeof(float));
    M2 = (float *)malloc(n * p * sizeof(float));

    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);

    Mout = (float *)malloc(n * p * sizeof(float));

    hipMalloc((void **)&d_M1, n * p * sizeof(float));
    hipMalloc((void **)&d_M2, n * p * sizeof(float));
    hipMalloc((void **)&d_Mout, n * p * sizeof(float));

    hipMemcpy(d_M1, M1, n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2, n * p * sizeof(float), hipMemcpyHostToDevice);

    clock_t start_time_add = clock();
    cudaMatrixAdd<<<n, p>>>(d_M1, d_M2, d_Mout, n, p);
    clock_t end_time_add = clock();

    double execution_time_add = ((double)end_time_add - start_time_add) / CLOCKS_PER_SEC;
    printf("Execution time for MatrixAdd: %f seconds\n", execution_time_add);


    hipMemcpy(Mout, d_Mout, n * p * sizeof(float), hipMemcpyDeviceToHost);

    //MatrixPrint(M1, n, p);
    //MatrixPrint(M2, n, p);

    //MatrixPrint(Mout, n, p);

    free(M1);
    free(M2);
    free(Mout);

    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);



    // cudaMatrixMult test

    n = 100;
    float *M3, *M4, *Mout2;
    float *d_M3, *d_M4, *d_Mout2;

    M3 = (float *)malloc(n * n * sizeof(float));
    M4 = (float *)malloc(n * n * sizeof(float));

    MatrixInit(M3, n, n);
    MatrixInit(M4, n, n);

    Mout2 = (float *)malloc(n * n * sizeof(float));

    hipMalloc((void **)&d_M3, n * n * sizeof(float));
    hipMalloc((void **)&d_M4, n * n * sizeof(float));
    hipMalloc((void **)&d_Mout2, n * n * sizeof(float));

    hipMemcpy(d_M3, M3, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M4, M4, n * n * sizeof(float), hipMemcpyHostToDevice);

    clock_t start_time_mult = clock();
    cudaMatrixMult<<<n, n>>>(d_M3, d_M4, d_Mout2, n);
    clock_t end_time_mult = clock();

    double execution_time_mult = ((double)end_time_mult - start_time_mult) / CLOCKS_PER_SEC;
    printf("Execution time for MatrixMult: %f seconds\n", execution_time_mult);

    hipMemcpy(Mout2, d_Mout2, n * n * sizeof(float), hipMemcpyDeviceToHost);

    //MatrixPrint(M3, n, n);
    //MatrixPrint(M4, n, n);

    //MatrixPrint(Mout2, n, n);

    free(M3);
    free(M4);
    free(Mout2);

    hipFree(d_M3);
    hipFree(d_M4);
    hipFree(d_Mout2);

    return 0;

}


/////////////////////// Layer 1 - Génération des données de test //////////////////////////
////////////////////////////////////// TEST RAW DATA //////////////////////////////////////


int test_raw_data() {

///////// init the raw_data matrix with random values between 0 and 1 ///////////
    
    int n = 32;
    float* raw_data = (float*)malloc(n * n * sizeof(float));
    float *d_M_raw;
    MatrixInit(raw_data, n, n);   

    hipMalloc((void **)&d_M_raw, n * n * sizeof(float));
    hipMemcpy(d_M_raw, raw_data, n * n * sizeof(float), hipMemcpyHostToDevice);
    // MatrixPrint(raw_data, n, n);

    free(raw_data);
    hipFree(d_M_raw);

///////// init the C1_data matrix with random values 0  /////////////////////////
     
    int c = 28;
    float* C1_data = (float*)malloc(6* c * c * sizeof(float));
    float *d_M_C1;
    int i, j, y;
    
    for (y = 0; y < 6; y++)
        for (i = 0; i < c; i++)
            for (j = 0; j < c; j++)
                C1_data[i * c + j] = static_cast<float>(0) ;


    hipMalloc((void **)&d_M_C1, 6 * c * c * sizeof(float));
    hipMemcpy(d_M_C1, C1_data, 6 * c * c * sizeof(float), hipMemcpyHostToDevice);
    // MatrixPrint(C1_data, 6*c, 6*c);

    free(C1_data);
    hipFree(d_M_C1);

///////// init the S1_data matrix with random values 0  /////////////////////////
 
    int d = 14;
    float* S1_data = (float*)malloc(d * d * sizeof(float));
    float *d_M_S1;

    for (y = 0; y < 6; y++)
        for (i = 0; i < d; i++)
            for (j = 0; j < d; j++)
                C1_data[i * d + j] = static_cast<float>(0) ;

    hipMalloc((void **)&d_M_S1, 6 * d * d * sizeof(float));
    hipMemcpy(d_M_S1, S1_data, 6 * d * d * sizeof(float), hipMemcpyHostToDevice);
    // MatrixPrint(S1_data, 6*d, 6*d);

    free(S1_data);
    hipFree(d_M_S1);


///////// init the C1_kernel matrix with random values 0  /////////////////////////
 
    int r = 5;
    float* C1_kernel = (float*)malloc(r * r * sizeof(float));
    float *d_M_C1_kernel;

    for (y = 0; y < 6; y++)
        for (i = 0; i < r; i++)
            for (j = 0; j < r; j++)
                C1_kernel[i * r + j] = (static_cast<float>(rand()) / RAND_MAX) * 2  ;

    hipMalloc((void **)&d_M_C1_kernel, 6 * r * r * sizeof(float));
    hipMemcpy(d_M_C1_kernel, C1_kernel, 6 * r * r * sizeof(float), hipMemcpyHostToDevice);
    // MatrixPrint(C1_kernel, 6*r, 6*r);

    free(C1_kernel);
    hipFree(d_M_C1_kernel);



//////////////////////////////////////////////////////////////////////////////////
//////////////////////////// Layer 2 - Convolution 2D ////////////////////////////





    return 0;
}




int main() {
    printf("Execution Time when using CPU\n");
    CPU_test();
    printf("Execution Time when using GPU\n");
    GPUtest();
    
    // test_raw_data();
    // conv_test();
    return 0;
}







