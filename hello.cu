#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
 
// __global__ keyword specifies a device kernel function
__global__ void kernelA(){
    printf("Hello, from the GPU!\n");
}
 
void MatrixInit(float *M, int n, int p){
    for(int i = 0; i < n; i++){
        for(int j = 0; j < p; j++){
            M[i*p + j] = (float)rand()/(float)RAND_MAX;
        }
    }
}

void MatrixPrint(float *M, int n, int p){
    for(int i = 0; i < n; i++){
        for(int j = 0; j < p; j++){
            printf("%f ", M[i*p + j]);
        }
        printf("\n");
    }
}

int main()
{
    printf("Hello, from the CPU!\n");
     
    // Set which device should be used
    // The code will default to 0 if not called though
    hipSetDevice(0);
 
    // Call a device function from the host: a kernel launch
    // Which will print from the device
    kernelA <<<1,1>>>();


    hipMalloc(void **devPtr, size_t count);
    // This call waits for all of the submitted GPU work to complete
    
    
    hipDeviceSynchronize();
 
   return 0;
}
